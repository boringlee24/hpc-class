
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define N 512        
#define BLOCKSIZE 32

double CLOCK() {
        struct timespec t;
        clock_gettime(CLOCK_MONOTONIC,  &t);
        return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void matmulKernel(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int idy = blockIdx.y*blockDim.y+threadIdx.y;
 
    // Make sure we do not go out of bounds
    if ((idx < n) && (idy < n))
    {
        double sum = 0;
        int k;
        for (k=0; k<n; k++)
        {
            sum += a[idy*n+k] * b[k*n+idx];
        }
        c[idy*n+idx] = sum;
    }        
}
 
int main( int argc, char* argv[] )
{
    double a[N][N]; /* input matrix */
    double b[N][N]; /* input matrix */
    double c[N][N]; /* result matrix */
    double *deviceA, *deviceB, *deviceC;
    int i,j;
    double start, finish, total;
    size_t size = N*N*sizeof(double);
    int n = N;

    /* initialize a dense matrix */
    for(i=0; i<N; i++){    
        for(j=0; j<N; j++){
        a[i][j] = (double)(i+j);
        b[i][j] = (double)(i-j);
        }
    }

    // allocate GPU memory space
    hipMalloc(&deviceA, size);
    hipMalloc(&deviceB, size);
    hipMalloc(&deviceC, size);

    start = CLOCK();

    // copy data
    hipMemcpy(deviceA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);
    dim3 dimGrid(16,16,1);

    matmulKernel<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, n);
 
    // Copy array back to host
    hipMemcpy(c, deviceC, size, hipMemcpyDeviceToHost);
 
    finish = CLOCK();

    printf("A result %g \n", c[7][8]);
    total = finish - start;
    printf("The total time for matrix multiplication with dense matrices = %f ms\n", total);
 
    /************************************************************************/
    // now do the sparse matrix multiply

    /* initialize a sparse matrix */
    int num_zeros = 0; 
    for(i=0; i<N; i++){    
        for(j=0; j<N; j++){
        if ((i<j)&&(i%2>0))
            {
            a[i][j] = (double)(i+j);
            b[i][j] = (double)(i-j);
            }
        else
            { 
            num_zeros++;
            a[i][j] = 0.0;
            b[i][j] = 0.0;
            }        
        }
    }    

    start = CLOCK();

    // copy data
    hipMemcpy(deviceA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, b, size, hipMemcpyHostToDevice);

    matmulKernel<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, n);
 
    // Copy array back to host
    hipMemcpy(c, deviceC, size, hipMemcpyDeviceToHost);

 
    finish = CLOCK();
    total = finish-start;
    printf("A result %g \n", c[7][8]); /* prevent dead code elimination */
    printf("The total time for matrix multiplication with sparse matrices = %f ms\n", total);
    printf("The sparsity of the a and b matrices = %f \n", (float)num_zeros/(float)(N*N));

    // Release device memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
 
    return 0;
}
