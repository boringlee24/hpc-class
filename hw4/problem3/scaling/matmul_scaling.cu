
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#define N 512        
#define BLOCKSIZE 32
#define GRIDSIZE 16

double CLOCK() {
        struct timespec t;
        clock_gettime(CLOCK_MONOTONIC,  &t);
        return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}
 
// CUDA kernel. Each thread takes care of one element of c
__global__ void matmulKernel(double *a, double *b, double *c, int n, int grid_size)
{
    // Get our global thread ID
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int idy = blockIdx.y*blockDim.y+threadIdx.y;
 
    // Make sure we do not go out of bounds
    if ((idx < n) && (idy < n))
    {
        int num_repeat = GRIDSIZE / grid_size;
        int i, j, k;
        for (i=0; i<num_repeat; i++)
        {
            for (j=0; j<num_repeat; j++)
            {
                double sum = 0;
                int new_idy = idy + BLOCKSIZE * grid_size * j;
                int new_idx = idx + BLOCKSIZE * grid_size * i;
                for (k=0; k<n; k++)
                {
                    sum += a[new_idy*n+k] * b[k*n+new_idx];
                }
                c[new_idy*n+new_idx] = sum;
            }
        }        
    }        
}
 
int main( int argc, char* argv[] )
{
    double a[N][N]; /* input matrix */
    double b[N][N]; /* input matrix */
    double c[N][N]; /* result matrix */
    double *deviceA, *deviceB, *deviceC;
    int i,j;
    double start, finish, total;
    size_t size = N*N*sizeof(double);
    int n = N;
    int grid_size = GRIDSIZE;

    if (argc == 2)
    {
        grid_size = strtol(argv[1], NULL, 10); 
        printf("Strong scaling test: grid dimension = (%d,%d) \n", grid_size, grid_size);
    }

    /* initialize a dense matrix */
    for(i=0; i<N; i++){    
        for(j=0; j<N; j++){
        a[i][j] = (double)(i+j);
        b[i][j] = (double)(i-j);
        }
    }

    // allocate GPU memory space
    hipMalloc(&deviceA, size);
    hipMalloc(&deviceB, size);
    hipMalloc(&deviceC, size);

    start = CLOCK();

    // copy data
    hipMemcpy(deviceA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, b, size, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCKSIZE, BLOCKSIZE, 1);
    dim3 dimGrid(grid_size,grid_size,1);

    matmulKernel<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, n, grid_size);
 
    // Copy array back to host
    hipMemcpy(c, deviceC, size, hipMemcpyDeviceToHost);
 
    finish = CLOCK();

    printf("A result %g \n", c[7][8]);
    total = finish - start;
    printf("The total time for matrix multiplication with dense matrices = %f ms\n", total);
 
    /************************************************************************/
    // now do the sparse matrix multiply

    /* initialize a sparse matrix */
    int num_zeros = 0; 
    for(i=0; i<N; i++){    
        for(j=0; j<N; j++){
        if ((i<j)&&(i%2>0))
            {
            a[i][j] = (double)(i+j);
            b[i][j] = (double)(i-j);
            }
        else
            { 
            num_zeros++;
            a[i][j] = 0.0;
            b[i][j] = 0.0;
            }        
        }
    }    

    start = CLOCK();

    // copy data
    hipMemcpy(deviceA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(deviceB, b, size, hipMemcpyHostToDevice);

    matmulKernel<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, n, grid_size);
 
    // Copy array back to host
    hipMemcpy(c, deviceC, size, hipMemcpyDeviceToHost);

 
    finish = CLOCK();
    total = finish-start;
    printf("A result %g \n", c[7][8]); /* prevent dead code elimination */
    printf("The total time for matrix multiplication with sparse matrices = %f ms\n", total);
    printf("The sparsity of the a and b matrices = %f \n", (float)num_zeros/(float)(N*N));

    // Release device memory
    hipFree(deviceA);
    hipFree(deviceB);
    hipFree(deviceC);
 
    return 0;
}
