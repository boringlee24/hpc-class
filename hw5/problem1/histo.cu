
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <math.h>
#include <time.h>

#define NUM_BIN 100
#define POWER 10
#define DATA_MAX 10000000
#define BLOCKSIZE 1024

using namespace std;

double CLOCK() {
        struct timespec t;
        clock_gettime(CLOCK_MONOTONIC,  &t);
        return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}

void init_histo(int* histogram) {
	int i;
	for (i = 0; i < NUM_BIN; i++)
	{
		histogram[i] = 0;
	}
}

void create_rand(int* data, int N) {
    int rand_gen, i;
	for (i = 0; i < N; i++)
	{
		rand_gen = rand() % DATA_MAX + 1;		
        data[i] = rand_gen;
	}
}

__global__ void histoKernel(int* data, int* histo, int data_size, int bin_size)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    int bin_idx, i;
    for (i = tid; i < data_size; i += (gridDim.x * blockDim.x))
    {
        bin_idx = (data[i] - 1) / bin_size;        
        atomicAdd(&histo[bin_idx], 1);
    }
}


int main (int argc, char** argv) {
    int histogram[NUM_BIN];
    double start, finish, total;
    int N = pow(2, POWER);
    printf("Number of datapoints: %d (2^%d), number of bins: %d \n", N, POWER, NUM_BIN);
    
    int data[N];
    int *device_data, *device_histo;

    init_histo(histogram);
    create_rand(data, N);

    size_t input_size = N*sizeof(int);
    size_t output_size = NUM_BIN*sizeof(int);

    // allocate GPU memory
    hipMalloc(&device_data, input_size);
    hipMalloc(&device_histo, output_size);

    int grid_dim = ceil(N/(float)BLOCKSIZE);
    dim3 threadPerBlock(BLOCKSIZE, 1, 1);
    dim3 blockPerGrid(grid_dim, 1, 1);
    int bin_size = DATA_MAX / NUM_BIN;
    printf("block size: %d, grid size: %d\n", BLOCKSIZE, grid_dim);

    start = CLOCK();

    // copy data
    hipMemcpy(device_data, data, input_size, hipMemcpyHostToDevice);
    hipMemcpy(device_histo, histogram, output_size, hipMemcpyHostToDevice);    

    // kernel launch
    histoKernel<<<blockPerGrid, threadPerBlock>>>(device_data, device_histo, N, bin_size);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    hipMemcpy(histogram, device_histo, output_size, hipMemcpyDeviceToHost);

    finish = CLOCK();
    total = finish - start;
    printf("The total time for histogram generation = %f ms\n", total);

    // Write the data out
    ofstream output_file;
    output_file.open("histogram.dat", ios::out | ios::trunc);
    for(int i = 0; i < NUM_BIN; i++){
        output_file << histogram[i] << " \n";
    }
    output_file.close();

    // Release device memory
    hipFree(device_data);
    hipFree(device_histo);

    return 0;
}