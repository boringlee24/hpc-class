
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define N 64
#define BLOCKSIZE 8

double CLOCK() {
        struct timespec t;
        clock_gettime(CLOCK_MONOTONIC,  &t);
        return (t.tv_sec * 1000)+(t.tv_nsec*1e-6);
}

__global__ void stencil(float* a, float* b)
{
    // Get our global thread ID
    int idx = blockIdx.x*blockDim.x+threadIdx.x;
    int idy = blockIdx.y*blockDim.y+threadIdx.y;
    int idz = blockIdx.z*blockDim.z+threadIdx.z;    

    // load into shared memory then sync threads
    __shared__ float temp[BLOCKSIZE][BLOCKSIZE][BLOCKSIZE];
    temp[threadIdx.x][threadIdx.y][threadIdx.z] = b[idx+N*(idy+N*idz)];
    __syncthreads();

    // use temp to calculate a
    if ((idx >= 1) && (idx < N-1) && (idy >= 1) && (idy < N-1) && (idz >= 1) && (idz < N-1))
    {
        float x_part = (threadIdx.x >= 1 ? temp[threadIdx.x-1][threadIdx.y][threadIdx.z] : b[(idx-1)+N*(idy+N*idz)]) + 
                        (threadIdx.x < BLOCKSIZE-1 ? temp[threadIdx.x+1][threadIdx.y][threadIdx.z] : b[(idx+1)+N*(idy+N*idz)]);
        float y_part = (threadIdx.y >= 1 ? temp[threadIdx.x][threadIdx.y-1][threadIdx.z] : b[idx+N*((idy-1)+N*idz)]) + 
                        (threadIdx.y < BLOCKSIZE-1 ? temp[threadIdx.x][threadIdx.y+1][threadIdx.z] : b[idx+N*((idy+1)+N*idz)]);
        float z_part = (threadIdx.z >= 1 ? temp[threadIdx.x][threadIdx.y][threadIdx.z-1] : b[idx+N*(idy+N*(idz-1))]) + 
                        (threadIdx.z < BLOCKSIZE-1 ? temp[threadIdx.x][threadIdx.y][threadIdx.z+1] : b[idx+N*(idy+N*(idz+1))]);                            
        a[idx+N*(idy+N*idz)] = 0.8 * (x_part + y_part + z_part);
    }
    else
    {
        a[idx+N*(idy+N*idz)] = 0;
    }
}


int main (int argc, char** argv) {
    float a[N][N][N], b[N][N][N];
    double start, finish, total;
    
    printf("input dimension: %dx%dx%d\n", N, N, N);
    
    float *device_a, *device_b;
    int i, j, k;

    /* initialize a dense matrix */
    for(i=0; i<N; i++){    
        for(j=0; j<N; j++){
            for(k=0; k<N; k++){
                b[i][j][k] = 1.25; //then a[][][] = 6
            }
        }
    }

    size_t size = N*N*N*sizeof(float);

    // allocate GPU memory
    hipMalloc(&device_a, size);
    hipMalloc(&device_b, size);

    int grid_dim = ceil(N/(float)BLOCKSIZE);
    dim3 threadPerBlock(BLOCKSIZE, BLOCKSIZE, BLOCKSIZE);
    dim3 blockPerGrid(grid_dim, grid_dim, grid_dim);
    printf("block size: %d^3, grid size: %d^3\n", BLOCKSIZE, grid_dim);


    // copy data
    // cudaMemcpy(device_a, a, size, cudaMemcpyHostToDevice);
    hipMemcpy(device_b, b, size, hipMemcpyHostToDevice);    

    // kernel launch
    start = CLOCK();
    stencil<<<blockPerGrid, threadPerBlock>>>(device_a, device_b);
    finish = CLOCK();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) 
        printf("Error: %s\n", hipGetErrorString(err));

    hipMemcpy(a, device_a, size, hipMemcpyDeviceToHost);

    total = finish - start;
    printf("The total time for stencil operation = %4.4f ms\n", total);

    printf("a[1][2][%d] = %f, a[8][8][8] = %f\n", N-1, a[1][2][N-1], a[8][8][8]);
    printf("expected values: 0 and 6\n");

    // Release device memory
    hipFree(device_a);
    hipFree(device_b);

    return 0;
}